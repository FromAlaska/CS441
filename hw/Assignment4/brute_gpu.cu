#include "hip/hip_runtime.h"
// Jim Samson
// CSF441 Computer Architecture
// Assignment 4 Part B
// This is the brute-force homework.

/***********************************************************************
 * brute_cpu.cu
 *
 * Brute force crack of an MD5 hash.  This program assumes the string key
 * is exactly 6 letters long and all uppercase letters (it is not too much
 * additional work to handle arbritrary symbols and a variable number of
 * letters).  
 *
 * It cracks a MD5 hash, currently hard-coded in main, by trying all
 * six letter strings from AAAAAA to ZZZZZZ, hashing each one, and checking
 * if it matches the target hash.
 *
 ***********************************************************************/
 #include <stdio.h>
 #include <string.h>
 #include <math.h>
 #include "md5.cu"
 
 // Convert a decimal number (starting at 0) to a corresponding 6 letter string
 // using base 26 to represent the string
 // s must be big enough to hold 6 chars plus a null (7 chars total)
 __device__ __host__ void intToString(int num, char *s) {
   int ones = (num) % 26;
   int twentySix = (num / 26) % 26;
   int twentySixSquared = (num / 26 / 26) % 26;
   int twentySixCubed = (num / 26 / 26 / 26) % 26;
   int twentySixFourth = (num / 26 / 26 / 26 / 26) % 26;
   int twentySixFifth = (num / 26 / 26 / 26 / 26 / 26) % 26;
   // Store appropriate char into the string
   int i = 0;
   s[i++] = twentySixFifth + 'A';
   s[i++] = twentySixFourth + 'A';
   s[i++] = twentySixCubed + 'A';
   s[i++] = twentySixSquared + 'A';
   s[i++] = twentySix + 'A';
   s[i++] = ones + 'A';
   s[i] = '\0';
 }
 
 // You may find this helpful for testing, this takes a 6 char string
 // like ABACAB and returns back the decimal number that maps to it
 // using the intToString function above
 int stringToInt(char *s) {
   int length = strlen(s);
   int sum = 0;
   int power = 0;
 
   for (int i = length-1; i >= 0; i--)
   {
   int digit = s[i] - 'A';
   sum += digit * pow(26,power);	
   power++;
   } 
   return sum;
 };
 
 __global__ void decrypt(int* result, uint32_t* md5Target) {
   uint32_t hash1, hash2, hash3, hash4;
   char key[7];
   uint8_t length = 6;

   for (int i = 0; i < 26*26*26*26; i++) {
     intToString(i+blockIdx.x*26*26*26*26*26+threadIdx.x*26*26*26*26, key); 

     md5Hash((unsigned char*) key, length, &hash1, &hash2, &hash3, &hash4);
     if ((hash1 == md5Target[0]) && (hash2 == md5Target[1]) && (hash3 == md5Target[2]) && (hash4 == md5Target[3])) {
          result[0] = i+blockIdx.x*26*26*26*26*26+threadIdx.x*26*26*26*26;
        }
   }
 };
 // Brute force search over the space of numbers 0 - 26^6, mapped to all 6 char 
 // uppercase strings. The resulting string is hashed using md5 and compared
 // to the target hash to see if it is the same. If so, we just cracked the
 // original string that produced the md5 target.
 int main()
 {
   // This is the md5 hash string we are trying to crack
   char md5_hash_string[] = "070d912366b1cf46a01aaf93c99f907d";
   int md5Target[4];  // The md5 hash string extracted into four integers
 
   

   // This loop extracts the md5 hash string into md5Target[0],[1],[2],[3]

   for(int i = 0; i < 4; i++) {
     char tmp[16];
     strncpy(tmp, md5_hash_string + i * 8, 8);
     sscanf(tmp, "%x", &md5Target[i]);
     md5Target[i] = (md5Target[i] & 0xFF000000) >> 24 | (md5Target[i] &
                  0x00FF0000) >> 8 | (md5Target[i] & 0x0000FF00) << 8 |
                 (md5Target[i] & 0x000000FF) << 24;
   }
 
   int *gpuResult;
   uint32_t *gpuMD5Target = (uint32_t *) malloc(sizeof(uint32_t));
   int result[1];

   hipMalloc((void **) &gpuMD5Target,sizeof(uint32_t)*4);
   hipMalloc((void **) &gpuResult, sizeof(int));
   hipMemcpy(gpuMD5Target, md5Target, 4*sizeof(uint32_t),hipMemcpyHostToDevice);
     
   decrypt<<<26,26>>>(gpuResult, gpuMD5Target);
   hipMemcpy(result,gpuResult, sizeof(int),hipMemcpyDeviceToHost);
   char key[7];
   intToString(result[0], key);

   printf("The Key is!!: %s \n",key);
   hipFree(gpuMD5Target);
   hipFree(gpuResult);
 }