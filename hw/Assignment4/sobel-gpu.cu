#include "hip/hip_runtime.h"
// Jim Samson
// CSF441 Computer Architecture 
// Assignment 4
// Most code is written by Dr. Mock
// This HW Assignment uses cuda and the Sobel filter to convert an image.

/***********************************************************************
 * sobel-cpu.cu
 *
 * Implements a Sobel filter on the image that is hard-coded in main.
 * You might add the image name as a command line option if you were
 * to use this more than as a one-off assignment.
 *
 * See https://stackoverflow.com/questions/17815687/image-processing-implementing-sobel-filter
 * or https://blog.saush.com/2011/04/20/edge-detection-with-the-sobel-operator-in-ruby/
 * for info on how the filter is implemented.
 *
 * Compile/run with:  nvcc sobel-cpu.cu -lfreeimage
 *
 ***********************************************************************/
 #include "FreeImage.h"
 #include "stdio.h"
 #include "math.h"
 
 
 #define threadsPerBlock 32
 // Returns the index into the 1d pixel array
 // Given te desired x,y, and image width
 __device__ int pixelIndex(int x, int y, int width)
 {
     return (y*width + x);
 }
 
 __global__ void sobel(char *returnPixels, int width, char *pixels)
 {
     int x = threadIdx.x + blockIdx.x * blockDim.x;
     int y = threadIdx.y + blockIdx.y * blockDim.y;
     
         int x00 = -1;  int x20 = 1;
         int x01 = -2;  int x21 = 2;
         int x02 = -1;  int x22 = 1;
         x00 *= pixels[pixelIndex(x-1,y-1,width)];
         x01 *= pixels[pixelIndex(x-1,y,width)];
         x02 *= pixels[pixelIndex(x-1,y+1,width)];
         x20 *= pixels[pixelIndex(x+1,y-1,width)];
         x21 *= pixels[pixelIndex(x+1,y,width)];
         x22 *= pixels[pixelIndex(x+1,y+1,width)];
         
         int y00 = -1;  int y10 = -2;  int y20 = -1;
         int y02 = 1;  int y12 = 2;  int y22 = 1;
         y00 *= pixels[pixelIndex(x-1,y-1,width)];
         y10 *= pixels[pixelIndex(x,y-1,width)];
         y20 *= pixels[pixelIndex(x+1,y-1,width)];
         y02 *= pixels[pixelIndex(x-1,y+1,width)];
         y12 *= pixels[pixelIndex(x,y+1,width)];
         y22 *= pixels[pixelIndex(x+1,y+1,width)];
 
         int px = x00 + x01 + x02 + x20 + x21 + x22;
         int py = y00 + y10 + y20 + y02 + y12 + y22;
         returnPixels[pixelIndex(x,y,width)] = sqrt(float(px*px + py*py));
 
 }
 
 int main()
 {
     FreeImage_Initialise();
     atexit(FreeImage_DeInitialise);
 
     // Load image and get the width and height
     FIBITMAP *image;
     image = FreeImage_Load(FIF_PNG, "coins.png", 0);
     if (image == NULL)
     {
         printf("Image Load Problem\n");
         exit(0);
     }
     int imgWidth;
     int imgHeight;
     imgWidth = FreeImage_GetWidth(image);
     imgHeight = FreeImage_GetHeight(image);
 
     // Convert image into a flat array of chars with the value 0-255 of the
     // greyscale intensity
     RGBQUAD aPixel;
     char *pixels;  
     int pixIndex = 0;
     pixels = (char *) malloc(sizeof(char)*imgWidth*imgHeight);
     for (int i = 0; i < imgHeight; i++)
      for (int j = 0; j < imgWidth; j++)
      {
        FreeImage_GetPixelColor(image,j,i,&aPixel);
        char grey = ((aPixel.rgbRed + aPixel.rgbGreen + aPixel.rgbBlue)/3);
        pixels[pixIndex++]=grey;
      }
 
     dim3 numThreads(threadsPerBlock, threadsPerBlock, 1);
     dim3 numberOfBlocks(ceil(imgWidth/threadsPerBlock), ceil(imgHeight/threadsPerBlock), 1);
     // Apply sobel operator to pixels, ignoring the borders
	 FIBITMAP *bitmap = FreeImage_Allocate(imgWidth, imgHeight, 24);
	 
	 // CUDA Sobel stuff
     char *devicePixels;
     char *devGPUPixels;
     char *hostReturn = (char *) malloc(sizeof(char)*imgWidth*imgHeight);
     hipMalloc((void**) &devicePixels, sizeof(char)*imgWidth*imgHeight);
     hipMalloc((void**) &devGPUPixels, sizeof(char)*imgWidth*imgHeight);
     hipMemcpy(devicePixels, pixels, sizeof(char)*imgWidth*imgHeight, hipMemcpyHostToDevice);
     hipMemcpy(devGPUPixels, hostReturn, sizeof(char)*imgWidth*imgHeight, hipMemcpyHostToDevice);
     sobel<<<numberOfBlocks, numThreads>>>(devGPUPixels, imgWidth, devicePixels);
	 hipMemcpy(hostReturn, devGPUPixels, sizeof(char)*imgWidth*imgHeight, hipMemcpyDeviceToHost);
	 
	 
     for (int i = 1; i < imgWidth-1; i++)
     {
       for (int j = 1; j < imgHeight-1; j++)
       {
         int sVal = float(hostReturn[j * imgWidth + i]);
         aPixel.rgbRed = sVal;
         aPixel.rgbGreen = sVal;
         aPixel.rgbBlue = sVal;
         FreeImage_SetPixelColor(bitmap, i, j, &aPixel);
       }
     }
     FreeImage_Save(FIF_PNG, bitmap, "coins-edg.png", 0);
   
     free(pixels);
     free(hostReturn);
     hipFree(devicePixels);
     hipFree(devGPUPixels);
     FreeImage_Unload(bitmap);
     FreeImage_Unload(image);
     return 0;
 }